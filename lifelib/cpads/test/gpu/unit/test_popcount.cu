#include "hip/hip_runtime.h"
#include <cpads/random/prng.hpp>
#include "gtest/gtest.h"

__global__ void popcount_kernel(uint32_t *a, int32_t *b) {

    allocTLSM(mem, uint32_t, 64, 3);

    mem[0] = blockIdx.x * blockDim.x + threadIdx.x;
    mem[1] = a[mem[0]];
    mem[2] = hh::popc32(mem[1]);
    b[mem[0]] = mem[2];

}

TEST(GPUIntrinsics, Popcount) {

    int n = 1000000;

    uint32_t *d_a;
    uint32_t *h_a;
    int32_t *d_b;
    int32_t *h_b;

    hipMalloc((void**) &d_a, 4 * n);
    hipMalloc((void**) &d_b, 4 * n);
    hipHostMalloc((void**) &h_a, 4 * n);
    hipHostMalloc((void**) &h_b, 4 * n);

    hh::PRNG pcg(1, 2, 3);

    for (int i = 0; i < n; i++) {
        h_a[i] = pcg.generate();
    }

    hipMemcpy(d_a, h_a, 4 * n, hipMemcpyHostToDevice);
    popcount_kernel<<<(n >> 6), 64>>>(d_a, d_b);
    hipMemcpy(h_b, d_b, 4 * n, hipMemcpyDeviceToHost);

    for (int i = 0; i < n; i++) {
        EXPECT_EQ(h_b[i], hh::popc32(h_a[i]));
    }

    hipFree(d_a);
    hipFree(d_b);
    hipHostFree(h_a);
    hipHostFree(h_b);

}
